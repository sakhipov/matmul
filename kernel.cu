#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define BLOCK_SIZE 32
__global__ void kernel_global(float *a, float *b, int n, float *c)
{
	int bx = blockIdx.x; // ����� ����� �� x
	int by = blockIdx.y; // ����� ����� �� y
	int tx = threadIdx.x; // ����� ���� � ����� �� x
	int ty = threadIdx.y; // ����� ���� � ����� �� y
	float sum = 0.0f;
	int ia = n * (BLOCK_SIZE * by + ty); // ����� ������ �� A�
	int ib = BLOCK_SIZE * bx + tx; // ����� ������� �� B�
	int ic = ia + ib; // ����� �������� �� ђ
	// ���������� �������� ������� C
	for (int k = 0; k < n; k++) sum += a[ia + k] * b[ib + k * n];
	c[ic] = sum;
}

int main()
{
	int N = 2048;
	int m, n, k;

	float timerValueGPU = 0.0f;
	float timerValueCPU = 0.0f;
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	int numBytes = N * N * sizeof(float);
	float* adev, *bdev, *cdev, *a, *b, *c, *cc;

	a = (float*)malloc(numBytes);
	b = (float*)malloc(numBytes);
	c = (float*)malloc(numBytes);
	cc = (float*)malloc(numBytes);

	for (n = 0; n < N; n++)
		for (m = 0; m < N; m++) {
			a[m + n * N] = 2.0f * m + n;
			b[m + n * N] = 1;
		}

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);

	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);



	hipEventRecord(start, 0);

	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

	kernel_global << < blocks, threads >> > (adev, bdev, N, cdev);
	err = hipPeekAtLastError();
	if (err != hipSuccess)
		printf(hipGetErrorString(err));
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time %f msec\n", timerValueGPU);



	hipEventRecord(start, 0);

	clock_t startc;
	clock_t stopc;

	startc = clock();

	for (n = 0; n < N; n++)
		for (m = 0; m < N; m++) {
			cc[m + n * N] = 0.f;
			for (k = 0; k < N; k++)
				cc[m + n * N] += a[k + n * N] * b[k + m * N];
		}

	stopc = clock();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueCPU, start, stop);
	printf("\n CPU calculation time %f msec\n", timerValueCPU);
	printf("\n Ctime: %f\n", ((double)(stopc - startc)) / ((double)CLOCKS_PER_SEC));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(&adev);
	hipFree(&bdev);
	hipFree(&cdev);

	delete a;
	delete b;
	delete c;
	delete cc;

	return 0;
}